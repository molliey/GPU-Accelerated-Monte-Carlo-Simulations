#include "hip/hip_runtime.h"
#include "simulate_gpu.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <vector>
#include <iostream>

__global__ void monteCarloCombinedKernel(float* d_returns, int numPaths, float mean, float stdDev, hiprandState* states, unsigned long seed) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= numPaths) return;

    // Initialize cuRAND state and generate a random sample
    hiprand_init(seed, tid, 0, &states[tid]);
    d_returns[tid] = mean + stdDev * hiprand_normal(&states[tid]);
}

float calculateVaRGPU(float* d_returns, int numPaths, float confidenceLevel) {
    // Wrap the raw pointer in a Thrust device vector
    thrust::device_vector<float> d_vec(d_returns, d_returns + numPaths);

    // Sort the vector on the GPU using Thrust
    thrust::sort(d_vec.begin(), d_vec.end());

    // Calculate the VaR (5th percentile for 95% confidence level)
    int index = static_cast<int>((1.0 - confidenceLevel) * numPaths);
    return -d_vec[index];
}

void simulateReturnsGPU(int numPaths, float mean, float stdDev, float* d_returns, hiprandState* d_states) {
    int blockSize = 512;
    int gridSize = (numPaths + blockSize - 1) / blockSize;

    unsigned long seed = clock();
    monteCarloCombinedKernel<<<gridSize, blockSize>>>(d_returns, numPaths, mean, stdDev, d_states, seed);

    // Synchronize to ensure the kernel has finished
    hipDeviceSynchronize();
}
