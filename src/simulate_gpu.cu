#include "hip/hip_runtime.h"
#include "simulate_gpu.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <vector>
#include <iostream>

__global__ void monteCarloKernel(float* d_returns, float mean, float stdDev, int numPaths) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= numPaths) return;

    hiprandState state;
    hiprand_init(clock64(), tid, 0, &state);
    d_returns[tid] = mean + stdDev * hiprand_normal(&state);
}

std::vector<float> simulateReturnsGPU(int numPaths, float mean, float stdDev) {
    float* d_returns;
    hipError_t err;
    
    err = hipMalloc(&d_returns, numPaths * sizeof(float));
    if (err != hipSuccess) {
        std::cerr << "CUDA malloc failed: " << hipGetErrorString(err) << std::endl;
        return std::vector<float>();
    }

    int blockSize = 256;
    int gridSize = (numPaths + blockSize - 1) / blockSize;
    
    monteCarloKernel<<<gridSize, blockSize>>>(d_returns, mean, stdDev, numPaths);
    
    hipDeviceSynchronize();
    err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA launch failed: " << hipGetErrorString(err) << std::endl;
        hipFree(d_returns);
        return std::vector<float>();
    }

    std::vector<float> h_returns(numPaths);
    err = hipMemcpy(h_returns.data(), d_returns, numPaths * sizeof(float), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        std::cerr << "CUDA memcpy failed: " << hipGetErrorString(err) << std::endl;
        hipFree(d_returns);
        return std::vector<float>();
    }

    hipFree(d_returns);

    return h_returns;
}
