#include "hip/hip_runtime.h"
#include "simulate_gpu.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <vector>
#include <iostream>

__global__ void monteCarloKernel(float* d_returns, int numPaths, float mean, float stdDev, hiprandState* states) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= numPaths) return;

    hiprandState localState = states[tid];
    d_returns[tid] = mean + stdDev * hiprand_normal(&localState);
    states[tid] = localState; 
}

__global__ void initializeCurandStates(hiprandState* states, unsigned long seed, int numPaths) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= numPaths) return;

    hiprand_init(seed, tid, 0, &states[tid]);
}

std::vector<float> simulateReturnsGPU(int numPaths, float mean, float stdDev) {
    float* d_returns;
    hipMalloc(&d_returns, numPaths * sizeof(float));

    hiprandState* d_states;
    hipMalloc(&d_states, numPaths * sizeof(hiprandState));

    int blockSize = 256;
    int gridSize = (numPaths + blockSize - 1) / blockSize;

    initializeCurandStates<<<gridSize, blockSize>>>(d_states, clock(), numPaths);

    monteCarloKernel<<<gridSize, blockSize>>>(d_returns, numPaths, mean, stdDev, d_states);

    std::vector<float> h_returns(numPaths);
    hipMemcpy(h_returns.data(), d_returns, numPaths * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_states);
    hipFree(d_returns);

    return h_returns;
}
