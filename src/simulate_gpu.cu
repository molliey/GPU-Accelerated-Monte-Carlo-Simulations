#include "hip/hip_runtime.h"
#include "simulate_gpu.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <vector>

__global__ void monteCarloKernel(float* d_returns, float mean, float stdDev, int numPaths) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= numPaths) return;

    hiprandState state;
    hiprand_init(1234, tid, 0, &state);

    d_returns[tid] = mean + stdDev * hiprand_normal(&state);
}

std::vector<float> simulateReturnsGPU(float mean, float stdDev, int numPaths) {
    float* d_returns;
    hipMalloc(&d_returns, numPaths * sizeof(float));

    int blockSize = 256;
    int gridSize = (numPaths + blockSize - 1) / blockSize;
    monteCarloKernel<<<gridSize, blockSize>>>(d_returns, mean, stdDev, numPaths);

    std::vector<float> h_returns(numPaths);
    hipMemcpy(h_returns.data(), d_returns, numPaths * sizeof(float), hipMemcpyDeviceToHost);
    hipFree(d_returns);

    return h_returns;
}
